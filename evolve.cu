
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define real double
#define CHECK_ERROR(cmd) if(cudaStatus=cmd != hipSuccess) printf("Erorr %s\n",hipGetErrorString(cudaStatus)) 

typedef struct Particle {
    real x;
    real y;
    real z;
    real vx;
    real vy;
    real vz;
    real dt;
    real energy;
} Particle;

__global__ void evolve(Particle *p, real tend, int n, real *params); 
void output(int time, Particle *p, int n) ;
__device__ void dy_potential(real x, real y, real *params, real *res) ;
__device__ void dx_potential(real x, real y, real *params, real *res) ;
__device__ void potential(real x, real y, real *params, real *res) ;
 void potential_cpu(real x, real y, real *params, real *res) ;
__device__ void leapfrog_step(Particle *p, real *params) ;
void set_particle_dt(Particle *p); 
void set_particle_ic(Particle *p,int n,real *params);
__device__ void set_energy(Particle *p,real *params);
 void set_energy_cpu(Particle *p,real *params);
int main(int argc, char *argv[]) {
    hipError_t cudaStatus;
    size_t size; 
    real params[2]={1.0,1.0};
    int ntot, nt;
    ntot = atoi(argv[1]);
    nt = atoi(argv[2]);
    params[0] = atof(argv[3]);
    params[1] = atof(argv[4]);
    
    int n = (int)sqrt(ntot);
    ntot = n*n;
    size = ntot*sizeof(Particle);

//    dim3 threadsPerBlock(8,8);
//    dim3 numBlocks(n/threadsPerBlock.x, n/threadsPerBlock.y);
    int threadsPerBlock=256;
    int blocksPerGrid = (ntot + threadsPerBlock-1)/threadsPerBlock;

    printf("Using NTOT=%d\tnt=%d\tq^2=%f\tR^2=%f\n",ntot,nt,params[0],params[1]);
    printf("Using %d blocksPerGrid, %d threadsPerBlock\n",threadsPerBlock,blocksPerGrid);

    Particle *particles;
    particles = (Particle *)malloc(size);

    Particle *particles_dev;
    hipMalloc((void **)&particles_dev, size);
    

    int j;
    real dt = 1.;

    set_particle_ic(particles,n,params);
    output(0,particles,n);
    
    hipMemcpy(particles_dev,particles,size,hipMemcpyHostToDevice);
    for(j=1;j<nt;j++) {
        evolve<<<blocksPerGrid,threadsPerBlock>>>(particles_dev,j*dt,n,params);
        cudaStatus=hipGetLastError();
        if (cudaStatus != hipSuccess) printf("%s\n",hipGetErrorString(cudaStatus));
        hipMemcpy(&particles[0],particles_dev,size,hipMemcpyDeviceToHost);
        output(j,particles,n);
    }
    free(particles);
    hipFree(particles_dev);
    return 1;
}


__global__ void evolve(Particle *p, real tend,int n,real *params) {
    int i,j,indx;
    real t=0;

    real dxp, dyp, pot;
    real R2 = params[1];
    real q2 = params[0];

    indx = threadIdx.x + blockIdx.x * blockDim.x;
    //j = threadIdx.y + blockIdx.y * blockDim.y;
   // indx = j + i*n;
    
    //if ( i<n && j<n) {
     
    if (indx < n*n){
//            p[indx].vx = 10;
//      p[indx].dt  = 1;
    //    while (t <= tend) {
            p[indx].x += p[indx].vx*p[indx].dt*.5;
            p[indx].y += p[indx].vy*p[indx].dt*.5;
            dx_potential(p[indx].x,p[indx].y,params,&dxp);
            dy_potential(p[indx].x,p[indx].y,params,&dyp);
            dxp = 2*p[indx].x/(R2 + p[indx].x*p[indx].x + p[indx].y*p[indx].y/q2); 
            dyp = 2*p[indx].y/(q2*(R2 + p[indx].x*p[indx].x) + p[indx].y*p[indx].y); 
            p[indx].vx += -p[indx].dt*dxp;
            p[indx].vy += -p[indx].dt*dyp;
            p[indx].x += p[indx].vx*p[indx].dt*.5;
            p[indx].y += p[indx].vy*p[indx].dt*.5;
            potential(p[indx].x,p[indx].y,params,&pot);
            pot = log(R2 + p[indx].x*p[indx].x + p[indx].y*p[indx].y/q2); 
            p[indx].energy = .5*(p[indx].vx*p[indx].vx +p[indx].vy*p[indx].vy) + pot;
            t += p[indx].dt;
      //  }
    }
    
    return;
}

__device__ void set_energy(Particle *p,real *params) {
    real res;
    potential(p->x,p->y,params,&res);
    p->energy = .5*(p->vx*p->vx + p->vy*p->vy + p->vz*p->vz);
    p->energy += res;
    return;
}
void set_energy_cpu(Particle *p,real *params) {
    real res;
    potential_cpu(p->x,p->y,params,&res);
    p->energy = .5*(p->vx*p->vx + p->vy*p->vy + p->vz*p->vz);
    p->energy += res;
    return;
}
__device__ void leapfrog_step(Particle *p,real *params) {

    real dt = p->dt;
    real x = p->x;
    real y = p->y;
    real vx = p->vx;
    real vy = p->vy;
    real dxp, dyp;

    x += vx*dt*.5;
    y += vy*dt*.5;
    dx_potential(x,y,params,&dxp);
    dy_potential(x,y,params,&dyp);

    vx += -dt*dxp;
    vy += -dt*dyp; 
    x += vx*dt*.5;
    y += vy*dt*.5;
    p->x = x;
    p->y = y;
    p->vx = vx;
    p->vy = vy;

    return;
}

__device__ void potential(real x, real y, real *params,real *res) {
    real q2 = params[0];
    real R2 = params[1];

    *res = log(x*x + y*y/q2 + R2);
    return;
}
 void potential_cpu(real x, real y, real *params,real *res) {
    real q2 = params[0];
    real R2 = params[1];

    *res = log(x*x + y*y/q2 + R2);
    return;
}
__device__ void dx_potential(real x, real y, real *params,real *res) {
    real q2 = params[0];
    real R2 = params[1];
    *res = 2*x/(R2 + x*x + y*y/q2); 
    return;
}
__device__ void dy_potential(real x, real y, real *params,real *res) {
    real q2 = params[0];
    real R2 = params[1];

    *res =  2*y/( q2*(R2 + x*x) + y*y); 
    return;
}
void set_particle_ic(Particle *p,int n,real *params) {

    int i,j;

    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            p[j+i*n].x = -6.5 + i*10.0/n;
            p[j+i*n].y = -6.5 + j*10.0/n;
            set_energy_cpu(&p[j+i*n],params);
            set_particle_dt(&p[j+i*n]);
        }
    }
    return;

}

void set_particle_dt(Particle *p) {
    p->dt = .1;
    return;
}

void output(int time, Particle *p,int n) {
    FILE *f;
    int i;
    char fname[100];
    sprintf(fname,"outputs/particles_%d.dat",time);
    f = fopen(fname,"w");

    for(i=0;i<n*n;i++) {
        fwrite(&p[i].x,sizeof(real),1,f);
        fwrite(&p[i].y,sizeof(real),1,f);
        fwrite(&p[i].z,sizeof(real),1,f);
        fwrite(&p[i].vx,sizeof(real),1,f);
        fwrite(&p[i].vy,sizeof(real),1,f);
        fwrite(&p[i].vz,sizeof(real),1,f);
        fwrite(&p[i].energy,sizeof(real),1,f);
//    fprintf(f,"%d\t%lg\t%lg\t%lg\t%lg\t%lg\t%lg\n",
//                i,p[i].x,p[i].y,p[i].z,p[i].vx,p[i].vy,p[i].vz);
    }
    fclose(f);
    return;

}
